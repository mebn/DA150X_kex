#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <private/job.h>
#include <string.h>
#include <iostream>
#include <set>

using namespace std;

set<string> createJob(job_t *job, std::map<std::string, std::string> elements)
{
    job->base.job_no = stoul(elements["EP"]);
    job->base.arriv_t = stod(elements["ARRIV_T"]);
    job->base.end_time = job->base.start_time = 0;
    job->base.qty = stoul(elements["QTY"]);

    job->r_qt = stod(elements["R_QT"]) * 60;
    job->recipe.ul_recipe = 0;
    strncpy(job->recipe.str_recipe, elements["RECIPE"].c_str(), 6);
    job->urgent = stod(elements["URGENT_W"]);
    // cout<<elements["CANRUN_TOOL"]<<endl;
    string can_run_tools = elements["CANRUN_TOOL"];
    string temp;
    set<string> v_tools;
    size_t start_pos = 0;
    do {
        temp = can_run_tools.substr(start_pos, 6);
        v_tools.insert(temp);
        start_pos += 6;
        cout << temp << endl;
    } while (start_pos != can_run_tools.length());
    job->base.size_of_process_time = v_tools.size();

    return v_tools;
}

__device__ __host__ double jobGetValue(void *_self)
{
    list_ele_t *self = (list_ele_t *) _self;
    job_t *j = (job_t *) self->ptr_derived_object;
    return *(j->base.os_seq_gene);
}

__device__ __host__ void initJob(job_t *self)
{
    _list_init(&self->list);
    self->list.ptr_derived_object = self;
    self->list.get_value = jobGetValue;

    job_base_init(&self->base);
    self->base.ptr_derived_object = self;
}
