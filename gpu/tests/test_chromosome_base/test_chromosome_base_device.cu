#include "hip/hip_runtime.h"
#include "include/common.h"
#include "include/job_base.h"
#include <bits/floatn.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <include/linked_list.h>
#include <include/machine_base.h>
#include <include/chromosome_base.h>
#include <tests/include/test_chromosome_base.h>
#include <tests/include/test_machine_base.h>
#include <tests/include/def.h>

size_t memory_usage = 0;

hipError_t test_cudaMalloc(void ** ptr, size_t size){
	memory_usage += size;
	return hipMalloc(ptr, size);
}

#define hipMalloc test_cudaMalloc 
		

extern int JOB_AMOUNT;
extern int MACHINE_AMOUNT;
extern int CHROMOSOME_AMOUNT;
extern int GENERATIONS;

class TestChromosomeBaseDevice : public testing::Test{
public:
	Machine ** machines;
	Machine ** address_machines_arr;
	job_t ** jobs;
	job_t ** address_jobs_arr;
	Chromosome * chromosomes;

	process_time_t **processTimes;
	process_time_t **address_process_time_arr;

	double **genes;
	double *host_genes;
	double **genes_address_arr;


	unsigned int *device_can_run_machine_size;
	unsigned int *host_can_run_machine_size;

	size_t gene_size;

	list_operations_t  *ops;
	machine_base_operations_t *mbops;
	job_base_operations_t *jbops;
	
	int R_JOB_AMOUNT;
	int R_MACHINE_AMOUNT;
	int R_CHROMOSOME_AMOUNT;

	void random_shuffle(double *genes, size_t size);

	void SetUp() override;
	void TearDown() override;
};

void TestChromosomeBaseDevice::random_shuffle(double *genes, size_t size){
	for(unsigned int i = 0; i < size; ++i){
		genes[i] = (double)rand() / (double)RAND_MAX;	
	}
}

void TestChromosomeBaseDevice::SetUp(){
	R_JOB_AMOUNT = JOB_AMOUNT * (CHROMOSOME_AMOUNT<<1);
	R_MACHINE_AMOUNT = MACHINE_AMOUNT * (CHROMOSOME_AMOUNT<<1);
	R_CHROMOSOME_AMOUNT = CHROMOSOME_AMOUNT << 1;
	
	// allocating jobs
	cudaCheck(hipMalloc((void**)&jobs, sizeof(job_t*) * R_CHROMOSOME_AMOUNT), "allocating jobs...");
	cudaCheck(hipHostMalloc((void**)&address_jobs_arr, sizeof(job_t*) * R_CHROMOSOME_AMOUNT), "allocating host address_job_arr");
	


	job_t * tmp;
	for(int i = 0; i < R_CHROMOSOME_AMOUNT; ++i){
		cudaCheck(hipMalloc((void**)&tmp, sizeof(job_t) * JOB_AMOUNT), "allocating jobs for a chromosome");
		address_jobs_arr[i] = tmp;
	}
	cudaCheck(hipMemcpy(jobs, address_jobs_arr, sizeof(job_t*) * R_CHROMOSOME_AMOUNT, hipMemcpyHostToDevice), "copy jobs from host to device");
	
	
	// allocating machines
	cudaCheck( hipMalloc((void**)&machines, sizeof(Machine*)*R_CHROMOSOME_AMOUNT), "alloating machines...");
	cudaCheck( hipHostMalloc((void**)&address_machines_arr, sizeof(Machine*)*R_CHROMOSOME_AMOUNT), "allocating host address_machines_arr");
	Machine *machines_tmp;
	for(int i = 0; i < R_CHROMOSOME_AMOUNT; ++i){
		cudaCheck( hipMalloc((void**)&machines_tmp, sizeof(Machine)*MACHINE_AMOUNT), "allocating machines for a chromosome");
		address_machines_arr[i] = machines_tmp;
	}
	cudaCheck( hipMemcpy(machines, address_machines_arr, sizeof(Machine*)*R_CHROMOSOME_AMOUNT, hipMemcpyHostToDevice), "copy machines from host to device");

	// allocating chromosomes
	cudaCheck( hipMalloc((void**)&chromosomes, sizeof(Chromosome)*R_CHROMOSOME_AMOUNT), "allocating chromosomes");

	
	// prepare host_can_run_machine_size
	cudaCheck( hipHostMalloc((void**)&host_can_run_machine_size, sizeof(unsigned int)*JOB_AMOUNT), "allocating host_can_run_machine_size on host");
	cudaCheck( hipMalloc((void**)&device_can_run_machine_size, sizeof(unsigned int)*JOB_AMOUNT), "allocating device_can_run_machines_size on device");
	for(int i = 0; i < JOB_AMOUNT; ++i){
		host_can_run_machine_size[i] = rand() % 200 + 400;
	}
	cudaCheck(hipMemcpy(device_can_run_machine_size, host_can_run_machine_size, sizeof(unsigned int)*JOB_AMOUNT, hipMemcpyHostToDevice), "copy can run tool");

	// prepare process_time
	cudaCheck( hipHostMalloc((void**)&address_process_time_arr, sizeof(process_time_t *)*JOB_AMOUNT), "allocating process time on host");
	cudaCheck( hipMalloc((void**)&processTimes, sizeof(process_time_t *)*JOB_AMOUNT), "allocating process time on device");
	process_time_t *process_time_tmp_host;
	process_time_t *process_time_tmp;
	for(int i = 0; i < JOB_AMOUNT; ++i){
		cudaCheck(hipMalloc((void**)&process_time_tmp, sizeof(process_time_t) * host_can_run_machine_size[i]), "allocating process time on device");
		cudaCheck(hipHostMalloc((void**)&process_time_tmp_host, sizeof(process_time_t) * host_can_run_machine_size[i]), "allocating process time on host");

		for(unsigned int j = 0; j < host_can_run_machine_size[i]; ++j){
			process_time_tmp_host[j].machine_no = rand() % MACHINE_AMOUNT;
			process_time_tmp_host[j].process_time = rand() % 1000;
		}

		cudaCheck(hipMemcpy(process_time_tmp, process_time_tmp_host, sizeof(process_time_t) * host_can_run_machine_size[i], hipMemcpyHostToDevice), "copy process time from host to deivce");
		cudaCheck(hipHostFree(process_time_tmp_host), "cuda free process_time_tmp_host");
		address_process_time_arr[i] = process_time_tmp;	
	}
	cudaCheck( hipMemcpy(processTimes, address_process_time_arr, sizeof(process_time_t *)*JOB_AMOUNT, hipMemcpyHostToDevice), "copy can run tool from host to device");

	// alloc genes
	double *genes_tmp;
	double *genes_host_tmp;
	cudaCheck(hipHostMalloc((void**)&genes_host_tmp, sizeof(double)*(JOB_AMOUNT<<1)), "hipHostMalloc for genes_host_tmp");
	cudaCheck(hipHostMalloc((void**)&genes_address_arr, sizeof(double*)*R_CHROMOSOME_AMOUNT), "hipHostMalloc for genes_address_arr");
	for(int i = 0; i < R_CHROMOSOME_AMOUNT; ++i){
		random_shuffle(genes_host_tmp, JOB_AMOUNT<<1);
		cudaCheck(hipMalloc((void**)&genes_tmp, sizeof(double)*(JOB_AMOUNT<<1)), "alloc for genes_tmp");
		cudaCheck(hipMemcpy(genes_tmp, genes_host_tmp, sizeof(double)*(JOB_AMOUNT<<1), hipMemcpyHostToDevice), "hipMemcpy for genes_tmp");
		genes_address_arr[i] = genes_tmp;
	}
	cudaCheck(hipMalloc((void**)&genes, sizeof(double*)*(R_CHROMOSOME_AMOUNT)),"cuda alloc genes");
	cudaCheck(hipHostFree(genes_host_tmp), "hipHostFree genes_host_tmp");
	cudaCheck(hipMemcpy(genes, genes_address_arr, sizeof(double*)*R_CHROMOSOME_AMOUNT, hipMemcpyHostToDevice), "hipMemcpy genes, genes_address_arr");

	// alloc ops
	cudaCheck(hipMalloc((void**)&ops, sizeof(list_operations_t)), "alloc ops");
	cudaCheck(hipMalloc((void**)&mbops, sizeof(machine_base_operations_t)), "alloc mbops");
	cudaCheck(hipMalloc((void**)&jbops, sizeof(job_base_operations_t)), "alloc jbops");
}

void TestChromosomeBaseDevice::TearDown(){
	// free ops
	cudaCheck(hipFree(ops), "Free ops...");
	cudaCheck(hipFree(mbops), "Free mbops");
	cudaCheck(hipFree(jbops), "Free jbops");

	// free jobs
	cudaCheck(hipFree(jobs), "Free jobs");
	for(int i = 0; i < R_CHROMOSOME_AMOUNT; ++i){
		cudaCheck(hipFree(address_jobs_arr[i]), "Free an array of jobs");
	}
	cudaCheck( hipHostFree(address_jobs_arr), "Free address_job_arr");

	// free machines
	cudaCheck(hipFree(machines), "Free machines");
	for(int i = 0; i < R_CHROMOSOME_AMOUNT; ++i){
		cudaCheck(hipFree(address_machines_arr[i]), "Free an array of machines");
	}
	cudaCheck(hipHostFree(address_machines_arr), "Free addres_machines_arr");

	// free can_run_machine
	cudaCheck(hipFree(device_can_run_machine_size), "Free device_can_run_machine_size");
	cudaCheck(hipHostFree(host_can_run_machine_size), "Free host_can_run_mahcine_size");

	// free chromosomes
	cudaCheck(hipFree(chromosomes), "Free chromosomes");

	// free process time
	cudaCheck(hipFree(processTimes), "Free processTimes");
	for(int i = 0; i < JOB_AMOUNT; ++i){
		cudaCheck(hipFree(address_process_time_arr[i]), "Free an array of process_time");
	}
	cudaCheck(hipHostFree(address_process_time_arr), "Free address_process_time_arr");
}

__global__ void machineSetup(Machine **machines, int MACHINE_AMOUNT, int CHROMOSOME_AMOUNT){
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y; 
	if(x < CHROMOSOME_AMOUNT && y < MACHINE_AMOUNT){
		// machines[x][y].base.init = initMachineBase;
		initMachine(&machines[x][y]);
	}
}

__global__ void chromosomeSetup(Chromosome *chromosomes, double ** genes, int JOB_AMOUNT, int CHROMOSOME_AMOUNT){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx < CHROMOSOME_AMOUNT){
		chromosomes[idx].val = idx;
		chromosomes[idx].base.gene_size = JOB_AMOUNT<<1;
		chromosomes[idx].base.chromosome_no = idx;
		chromosome_base_init(&chromosomes[idx].base, genes[idx]);
	}
}

__global__ void jobSetup(job_t ** jobs, unsigned int *can_run_tool_size, process_time_t ** process_times, job_base_operations_t *ops, int JOB_AMOUNT, int CHROMOSOME_AMOUNT){
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	if(x < CHROMOSOME_AMOUNT && y < JOB_AMOUNT){
		initJob(&jobs[x][y]);
		jobs[x][y].base.job_no = y;
		ops->set_process_time(&jobs[x][y].base, process_times[y], can_run_tool_size[y]);
		// jobs[x][y].base.setProcessTime(&jobs[x][y].base, process_times[y], can_run_tool_size[y]);
	}
}

__global__ void jobBindGenes(job_t **jobs, Chromosome * chromosomes, job_base_operations_t *jbops, int JOB_AMOUNT, int R_CHROMOSOME_AMOUNT){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y; 
	if(x < R_CHROMOSOME_AMOUNT && y < JOB_AMOUNT){
		jbops->set_ms_gene_addr(&jobs[x][y].base, chromosomes[x].base.ms_genes + y);
		jbops->set_os_gene_addr(&jobs[x][y].base, chromosomes[x].base.os_genes + y);
	}
}

__global__ void machineSelection(job_t **jobs, job_base_operations_t *jbops, int JOB_AMOUNT, int R_CHROMOSOME_AMOUNT){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int machine_idx;
	if(x < R_CHROMOSOME_AMOUNT && y < JOB_AMOUNT){
		machine_idx = jbops->machine_selection(&jobs[x][y].base);
		jobs[x][y].base.machine_no = jobs[x][y].base.process_time[machine_idx].machine_no;
	}
}

__global__ void machineSelection2(job_t **jobs, Machine **machines, machine_base_operations_t *ops, int JOB_AMOUNT, int MACHINE_AMOUNT, int R_CHROMOSOME_AMOUNT){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( x < R_CHROMOSOME_AMOUNT && y < MACHINE_AMOUNT){
		for(int i = 0; i < JOB_AMOUNT; ++i){
			if(jobs[x][i].base.machine_no == y){
				ops->add_job(&machines[x][y].base, &jobs[x][i].ele);
			}
		}
	}
}

__global__ void sortJob(Machine **machines, machine_base_operations_t *mbops, list_operations_t *ops, int MACHINE_AMOUNT, int R_CHROMOSOME_AMOUNT){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( x < R_CHROMOSOME_AMOUNT && y < MACHINE_AMOUNT){
	    mbops->sort_job(&machines[x][y].base, ops);
//		__sortJob(&machines[x][y].base, ops);
	}
}

__global__ void resetMachines(Machine **machines, machine_base_operations_t *mbops, int MACHINE_AMOUNT, int R_CHROMOSOME_AMOUNT){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( x < R_CHROMOSOME_AMOUNT && y < MACHINE_AMOUNT){
		mbops->reset(&machines[x][y].base);
	}
}

__global__ void operationSetup(list_operations_t *ops,job_base_operations_t *jbops,  machine_base_operations_t *mbops){
	ops->init = _list_init;
	ops->set_next = _list_ele_set_next;
	ops->set_prev = _list_ele_set_prev;
	
	machine_base_operations_t mbtmp = MACHINE_BASE_OPS;
	*mbops = mbtmp;

	job_base_operations_t jbtmp = JOB_BASE_OPS;
	*jbops = jbtmp;
}

TEST_F(TestChromosomeBaseDevice, test_chromosome_base_device){
	// setup grid dimension
	dim3 machine_chromosome_thread(2, 512);
	dim3 machine_chromosome_block(R_CHROMOSOME_AMOUNT >> 1, MACHINE_AMOUNT >> 8);
	
	dim3 job_chromosome_thread(2, 512);
	dim3 job_chromosome_block(R_CHROMOSOME_AMOUNT >> 1, JOB_AMOUNT >> 8); // (R_CHROMOSOME_AMOUNT / 32, )

	// setup kernel
	operationSetup<<<1, 1>>>(ops, jbops, mbops);
	jobSetup<<<job_chromosome_block, job_chromosome_thread>>>(jobs, device_can_run_machine_size, processTimes, jbops, JOB_AMOUNT, R_CHROMOSOME_AMOUNT);
	machineSetup<<<machine_chromosome_block, machine_chromosome_thread>>>(machines, MACHINE_AMOUNT, R_CHROMOSOME_AMOUNT);
	chromosomeSetup<<<100, 100>>>(chromosomes, genes, JOB_AMOUNT, R_CHROMOSOME_AMOUNT);

	jobBindGenes<<<job_chromosome_block, job_chromosome_thread>>>(jobs, chromosomes, jbops, JOB_AMOUNT, R_CHROMOSOME_AMOUNT);
	hipDeviceSynchronize();
	PRINTF("Device Memory Usage = %lu\n", memory_usage);

	hipEvent_t startEvent, stopEvent;
	cudaCheck(hipEventCreate(&startEvent), "create start event");
	cudaCheck(hipEventCreate(&stopEvent), "create stop event");

	// start computing...
	PRINTF("Start Computing...\n");
	cudaCheck(hipEventRecord(startEvent, 0), "cuda event record start event");
	for(int i = 0; i < GENERATIONS; ++i){
		machineSelection<<<job_chromosome_block, job_chromosome_thread>>>(jobs, jbops, JOB_AMOUNT, R_CHROMOSOME_AMOUNT);  // machine selection
		// PRINTF("Finish machine selection part 1\n");
		// PRINTF("Start machine selection part2\n");
		machineSelection2<<<machine_chromosome_block, machine_chromosome_thread>>>(jobs, machines, mbops, JOB_AMOUNT, MACHINE_AMOUNT, R_CHROMOSOME_AMOUNT);
		// PRINTF("Finish machine selection part2\n");
		sortJob<<<machine_chromosome_block, machine_chromosome_thread>>>(machines, mbops, ops, MACHINE_AMOUNT, R_CHROMOSOME_AMOUNT);
		resetMachines<<<machine_chromosome_block, machine_chromosome_thread>>>(machines, mbops, MACHINE_AMOUNT, R_CHROMOSOME_AMOUNT);
		hipDeviceSynchronize();
		// PRINTF("Finish sorting\n");
		PRINTF("Finish generation %d\n", i);
	 }
	
	cudaCheck(hipEventRecord(stopEvent, 0), "cuda event record stop event");
	cudaCheck(hipEventSynchronize(stopEvent), "cuda event sync stop event");

	float ms;
	cudaCheck(hipEventElapsedTime(&ms, startEvent, stopEvent), "get elapsed time");

	PRINTF("Elapsed Time : %.3fs\n", ms / 1000.0);
}
