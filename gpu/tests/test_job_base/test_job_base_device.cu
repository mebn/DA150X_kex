#include "hip/hip_runtime.h"
#include <cstring>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <include/job_base.h>
#include <vector>
#include <gtest/gtest.h>
#include <iostream>
#include <fstream>

#include <tests/include/test_job_base.h>

#define amount 100000

using namespace std;


class TestJobBaseDevice : public testing::Test{
protected:
	// job_t * jb;
	job_t ** jb_host;
	job_t ** device_jb_addresses;
	job_t ** jb_device;
	unsigned int * result_device;
	unsigned int * result_host;
	double arrayOfMsGene[amount];
	double * array_of_ms_gene_device;
	unsigned int arrayOfSizePt[amount], arrayOfMcNum[amount];
	void SetUp() override;
	void TearDown() override;
	void copyArrayOfJobBase(job_t **, job_t **);
	void setMsGeneData();
};

void TestJobBaseDevice::SetUp() {
	// initialize jb_host_*
	size_t sizeof_array_of_pointer = sizeof(job_t*) * amount;
	size_t sizeof_array_of_result = sizeof(unsigned int) * amount;

	// host memory allocation
	jb_host = (job_t **)malloc(sizeof_array_of_pointer);
	device_jb_addresses = (job_t **)malloc(sizeof_array_of_pointer);
	result_host = (unsigned int *)malloc(sizeof_array_of_result);

	// device memory allocation
	hipMalloc((void **)&jb_device, sizeof_array_of_pointer);
	hipMalloc((void **)&result_device, sizeof_array_of_result);

	setMsGeneData();
	// initializae host array
	for(unsigned int i = 0 ;i < amount; ++i){
		jb_host[i] = newJob(arrayOfSizePt[i]);	
		// jb_host[i] = new JobBaseChild(i);
		// jb_host[i]->set_ms_gene_addr(&arrayOfMsGene[i]);
		// jb_host[i]->setProcessTime(NULL, arrayOfSizePt[i]);
	}
	//initilize device array
	copyArrayOfJobBase(device_jb_addresses, jb_host);

	// copy content from host to device
	ASSERT_EQ(hipMemcpy(jb_device, device_jb_addresses, sizeof_array_of_pointer, hipMemcpyHostToDevice), hipSuccess);
}

void TestJobBaseDevice::copyArrayOfJobBase(job_t** device_address, job_t** src){
	job_t * device_temp_jb;
	size_t size = sizeof(job_t);
	for(unsigned int i = 0; i < amount; ++i){
		ASSERT_EQ(hipMalloc((void**)&device_temp_jb, size), hipSuccess);
		ASSERT_EQ(hipMemcpy(device_temp_jb, src[i], size, hipMemcpyHostToDevice), hipSuccess);
		device_address[i] = device_temp_jb;
	}
}


void TestJobBaseDevice::TearDown(){
}

void TestJobBaseDevice::setMsGeneData(){
    ifstream file;
    file.open("./ms_data.txt", ios::in);
    if (file){
		for(unsigned int i = 0; i < amount; ++i){
			file >> arrayOfMsGene[i] >> arrayOfSizePt[i] >> arrayOfMcNum[i];
		}
        
    }else {
        cout << "Unable to open file\n";
    }
    file.close();
}

__global__ void testMachineSelection(job_t ** jb, unsigned int * result, double * msgene_device, unsigned int numElements){
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	job_base_operations_t jbops = JOB_BASE_OPS;
	if(id < numElements){
		//jb[id]->base.init = job_base_init;
		//jb[id]->base.init(&jb[id]->base);
		job_base_init(&jb[id]->base);
		jbops.set_ms_gene_addr(&jb[id]->base, &(msgene_device[id]));
		result[id] = jbops.machine_selection(&jb[id]->base);
		// jb[id]->base.set_ms_gene_addr(&jb[id]->base, &(msgene_device[id]));
		// result[id] = jb[id]->base.machine_selection(&jb[id]->base);
	}
}


TEST_F(TestJobBaseDevice, test_machine_selection_host){
	job_base_operations_t jbops = JOB_BASE_OPS;
	for(int i = 0; i < amount; ++i){
		jbops.set_ms_gene_addr(&jb_host[i]->base, &arrayOfMsGene[i]);
		ASSERT_EQ(jbops.machine_selection(&jb_host[i]->base), arrayOfMcNum[i]) << "Entry : "<<i<<std::endl;
		// jb_host[i]->base.set_ms_gene_addr(&jb_host[i]->base, &arrayOfMsGene[i]);
		// ASSERT_EQ(jb_host[i]->base.machine_selection(&jb_host[i]->base), arrayOfMcNum[i]) << "Entry : "<<i<<std::endl;
	}	
}



TEST_F(TestJobBaseDevice, test_machine_selection_device){
	// copy the array content from host to device
	double * msgene_device;
	size_t size_arr = sizeof(double) * amount;
	ASSERT_EQ(hipMalloc((void**)&msgene_device, size_arr), hipSuccess);
	ASSERT_EQ(hipMemcpy(msgene_device, arrayOfMsGene, size_arr, hipMemcpyHostToDevice), hipSuccess);
	// computing
	testMachineSelection<<<256, 1024>>>(jb_device, result_device, msgene_device, amount);
	// copy the array content from device to host
	size_t size = sizeof(unsigned int) * amount;
	ASSERT_EQ(hipMemcpy(result_host, result_device, size, hipMemcpyDeviceToHost), hipSuccess);
	
	// testing
	for(unsigned int i = 0; i < amount; ++i){
		ASSERT_EQ(result_host[i], arrayOfMcNum[i]) << "Entry : " << i << std::endl;
	}
}

